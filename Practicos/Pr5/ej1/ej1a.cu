#include <iostream>
#include <vector>
#include <hip/hip_runtime.h>
#include <algorithm>
#include <chrono>
#include <numeric>
#include <cmath>

__global__ void block_scan_kernel(int* d_input, int* d_output, int* d_block_sums, int n) {
    extern __shared__ int temp[];
    int thid = threadIdx.x;
    int offset = 1;

    int block_start = 2 * blockIdx.x * blockDim.x;
    if (block_start + 2 * thid < n) {
        temp[2 * thid] = d_input[block_start + 2 * thid];
    } else {
        temp[2 * thid] = 0;
    }
    if (block_start + 2 * thid + 1 < n) {
        temp[2 * thid + 1] = d_input[block_start + 2 * thid + 1];
    } else {
        temp[2 * thid + 1] = 0;
    }
    __syncthreads();

    for (int d = blockDim.x; d > 0; d >>= 1) {
        if (thid < d) {
            int ai = offset * (2 * thid + 1) - 1;
            int bi = offset * (2 * thid + 2) - 1;
            temp[bi] += temp[ai];
        }
        offset *= 2;
        __syncthreads();
    }

    if (thid == 0) {
        d_block_sums[blockIdx.x] = temp[2 * blockDim.x - 1];
        temp[2 * blockDim.x - 1] = 0;
    }

    for (int d = 1; d < 2 * blockDim.x; d *= 2) {
        offset >>= 1;
        __syncthreads();
        if (thid < d) {
            int ai = offset * (2 * thid + 1) - 1;
            int bi = offset * (2 * thid + 2) - 1;
            int t = temp[ai];
            temp[ai] = temp[bi];
            temp[bi] += t;
        }
    }
    __syncthreads();

    if (block_start + 2 * thid < n) {
        d_output[block_start + 2 * thid] = temp[2 * thid];
    }
    if (block_start + 2 * thid + 1 < n) {
        d_output[block_start + 2 * thid + 1] = temp[2 * thid + 1];
    }
}

__global__ void add_block_sums_kernel(int* d_output, int* d_block_sums, int n) {
    int thid = threadIdx.x;
    int block_start = 2 * blockIdx.x * blockDim.x;
    if (blockIdx.x > 0) {
        if (block_start + 2 * thid < n) {
            d_output[block_start + 2 * thid] += d_block_sums[blockIdx.x];
        }
        if (block_start + 2 * thid + 1 < n) {
            d_output[block_start + 2 * thid + 1] += d_block_sums[blockIdx.x];
        }
    }
}

void exclusive_scan(const std::vector<int>& input, std::vector<int>& output) {
    int n = input.size();
    int* d_input = nullptr;
    int* d_output = nullptr;
    int* d_block_sums = nullptr;

    int blockSize = 512;
    int numBlocks = (n + 2 * blockSize - 1) / (2 * blockSize);

    hipMalloc(&d_input, n * sizeof(int));
    hipMalloc(&d_output, n * sizeof(int));
    hipMalloc(&d_block_sums, numBlocks * sizeof(int));
    hipMemcpy(d_input, input.data(), n * sizeof(int), hipMemcpyHostToDevice);

    int sharedMemorySize = 2 * blockSize * sizeof(int);

    block_scan_kernel<<<numBlocks, blockSize, sharedMemorySize>>>(d_input, d_output, d_block_sums, n);
    hipDeviceSynchronize();

    if (numBlocks > 1) {
        std::vector<int> block_sums(numBlocks);
        std::vector<int> block_sums_scan(numBlocks);
        hipMemcpy(block_sums.data(), d_block_sums, numBlocks * sizeof(int), hipMemcpyDeviceToHost);

        block_sums_scan[0] = 0;
        for (int i = 1; i < numBlocks; ++i) {
            block_sums_scan[i] = block_sums_scan[i - 1] + block_sums[i - 1];
        }

        hipMemcpy(d_block_sums, block_sums_scan.data(), numBlocks * sizeof(int), hipMemcpyHostToDevice);
        add_block_sums_kernel<<<numBlocks, blockSize>>>(d_output, d_block_sums, n);
        hipDeviceSynchronize();
    }

    hipMemcpy(output.data(), d_output, n * sizeof(int), hipMemcpyDeviceToHost);

    hipFree(d_input);
    hipFree(d_output);
    hipFree(d_block_sums);
}

void exclusive_scan_sequential(const std::vector<int>& input, std::vector<int>& output) {
    int n = input.size();
    output[0] = 0;
    for (int i = 1; i < n; ++i) {
        output[i] = output[i - 1] + input[i - 1];
    }
}

int main() { 

    //Warm-up
    std::vector<int> warmup_input(1024, 1);
    std::vector<int> warmup_output(1024);

    exclusive_scan(warmup_input, warmup_output);
    for (int k = 6; k <= 16; ++k) {
        int n = 1024*(1<<k);
        std::vector<int> input(n);
        std::vector<int> output_gpu(n);
        std::vector<int> output_cpu(n);

        for (int i = 0; i < n; ++i) {
            input[i] = i + 1;
        }

        std::vector<double> times;
        for (int i = 0; i < 10; ++i) {
            auto start = std::chrono::high_resolution_clock::now();
            exclusive_scan(input, output_gpu);
            auto end = std::chrono::high_resolution_clock::now();
            std::chrono::duration<double, std::milli> duration = end - start;
            times.push_back(duration.count());
        }

        exclusive_scan_sequential(input, output_cpu);

        bool are_equal = std::equal(output_gpu.begin(), output_gpu.end(), output_cpu.begin());

        double mean_time = std::accumulate(times.begin(), times.end(), 0.0) / times.size();
        double sq_sum = std::inner_product(times.begin(), times.end(), times.begin(), 0.0);
        double stddev_time = std::sqrt(sq_sum / times.size() - mean_time * mean_time);

        std::cout << "N = " << k << " -> " << (are_equal ? "Iguales" : "Diferentes") << "\t" << "Media de tiempo: " << mean_time << " ms\t" << "Desviacion: " << stddev_time << " ms\n";
    } 
    return 0;
}