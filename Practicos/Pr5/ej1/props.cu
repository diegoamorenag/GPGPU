#include <iostream>
#include <hip/hip_runtime.h>

int main() {
    int deviceCount;
    hipError_t err = hipGetDeviceCount(&deviceCount);
    if (err != hipSuccess) {
        std::cerr << "Error obteniendo el número de dispositivos CUDA: " << hipGetErrorString(err) << std::endl;
        return -1;
    }

    for (int device = 0; device < deviceCount; ++device) {
        hipDeviceProp_t deviceProp;
        err = hipGetDeviceProperties(&deviceProp, device);
        if (err != hipSuccess) {
            std::cerr << "Error obteniendo las propiedades del dispositivo: " << hipGetErrorString(err) << std::endl;
            return -1;
        }

        std::cout << "Propiedades del dispositivo " << device << ":" << std::endl;
        std::cout << "  Nombre: " << deviceProp.name << std::endl;
        std::cout << "  Memoria compartida por bloque: " << deviceProp.sharedMemPerBlock << " bytes" << std::endl;
        std::cout << "  Memoria compartida por multiprocesador: " << deviceProp.sharedMemPerMultiprocessor << " bytes" << std::endl;
        std::cout << "  Número de multiprocesadores: " << deviceProp.multiProcessorCount << std::endl;
    }

    return 0;
}