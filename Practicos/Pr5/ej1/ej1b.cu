#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>
#include <hip/hip_runtime.h>
#include <algorithm>
#include <chrono>
#include <numeric>
#include <cmath>
#include <hipcub/hipcub.hpp>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/scan.h>

// Custom exclusive scan implementation
__global__ void block_scan_kernel(int* d_input, int* d_output, int* d_block_sums, int n) {
    extern __shared__ int temp[];
    int thid = threadIdx.x;
    int offset = 1;

    int block_start = 2 * blockIdx.x * blockDim.x;
    if (block_start + 2 * thid < n) {
        temp[2 * thid] = d_input[block_start + 2 * thid];
    } else {
        temp[2 * thid] = 0;
    }
    if (block_start + 2 * thid + 1 < n) {
        temp[2 * thid + 1] = d_input[block_start + 2 * thid + 1];
    } else {
        temp[2 * thid + 1] = 0;
    }
    __syncthreads();

    for (int d = blockDim.x; d > 0; d >>= 1) {
        if (thid < d) {
            int ai = offset * (2 * thid + 1) - 1;
            int bi = offset * (2 * thid + 2) - 1;
            temp[bi] += temp[ai];
        }
        offset *= 2;
        __syncthreads();
    }

    if (thid == 0) {
        d_block_sums[blockIdx.x] = temp[2 * blockDim.x - 1];
        temp[2 * blockDim.x - 1] = 0;
    }

    for (int d = 1; d < 2 * blockDim.x; d *= 2) {
        offset >>= 1;
        __syncthreads();
        if (thid < d) {
            int ai = offset * (2 * thid + 1) - 1;
            int bi = offset * (2 * thid + 2) - 1;
            int t = temp[ai];
            temp[ai] = temp[bi];
            temp[bi] += t;
        }
    }
    __syncthreads();

    if (block_start + 2 * thid < n) {
        d_output[block_start + 2 * thid] = temp[2 * thid];
    }
    if (block_start + 2 * thid + 1 < n) {
        d_output[block_start + 2 * thid + 1] = temp[2 * thid + 1];
    }
}

__global__ void add_block_sums_kernel(int* d_output, int* d_block_sums, int n) {
    int thid = threadIdx.x;
    int block_start = 2 * blockIdx.x * blockDim.x;
    if (blockIdx.x > 0) {
        if (block_start + 2 * thid < n) {
            d_output[block_start + 2 * thid] += d_block_sums[blockIdx.x];
        }
        if (block_start + 2 * thid + 1 < n) {
            d_output[block_start + 2 * thid + 1] += d_block_sums[blockIdx.x];
        }
    }
}

void exclusive_scan(const std::vector<int>& input, std::vector<int>& output) {
    int n = input.size();
    int* d_input = nullptr;
    int* d_output = nullptr;
    int* d_block_sums = nullptr;

    int blockSize = 512;  // Using 512 threads per block for better performance
    int numBlocks = (n + 2 * blockSize - 1) / (2 * blockSize);

    hipMalloc(&d_input, n * sizeof(int));
    hipMalloc(&d_output, n * sizeof(int));
    hipMalloc(&d_block_sums, numBlocks * sizeof(int));
    hipMemcpy(d_input, input.data(), n * sizeof(int), hipMemcpyHostToDevice);

    int sharedMemorySize = 2 * blockSize * sizeof(int);

    block_scan_kernel<<<numBlocks, blockSize, sharedMemorySize>>>(d_input, d_output, d_block_sums, n);
    hipDeviceSynchronize();

    if (numBlocks > 1) {
        std::vector<int> block_sums(numBlocks);
        std::vector<int> block_sums_scan(numBlocks);
        hipMemcpy(block_sums.data(), d_block_sums, numBlocks * sizeof(int), hipMemcpyDeviceToHost);

        block_sums_scan[0] = 0;
        for (int i = 1; i < numBlocks; ++i) {
            block_sums_scan[i] = block_sums_scan[i - 1] + block_sums[i - 1];
        }

        hipMemcpy(d_block_sums, block_sums_scan.data(), numBlocks * sizeof(int), hipMemcpyHostToDevice);
        add_block_sums_kernel<<<numBlocks, blockSize>>>(d_output, d_block_sums, n);
        hipDeviceSynchronize();
    }

    hipMemcpy(output.data(), d_output, n * sizeof(int), hipMemcpyDeviceToHost);

    hipFree(d_input);
    hipFree(d_output);
    hipFree(d_block_sums);
}

void exclusive_scan_sequential(const std::vector<int>& input, std::vector<int>& output) {
    int n = input.size();
    output[0] = 0;
    for (int i = 1; i < n; ++i) {
        output[i] = output[i - 1] + input[i - 1];
    }
}

// CUB implementation of exclusive scan
void exclusive_scan_cub(const std::vector<int>& input, std::vector<int>& output) {
    int n = input.size();
    int* d_input = nullptr;
    int* d_output = nullptr;

    hipMalloc(&d_input, n * sizeof(int));
    hipMalloc(&d_output, n * sizeof(int));
    hipMemcpy(d_input, input.data(), n * sizeof(int), hipMemcpyHostToDevice);

    void* d_temp_storage = nullptr;
    size_t temp_storage_bytes = 0;
    hipcub::DeviceScan::ExclusiveSum(d_temp_storage, temp_storage_bytes, d_input, d_output, n);
    hipMalloc(&d_temp_storage, temp_storage_bytes);
    hipcub::DeviceScan::ExclusiveSum(d_temp_storage, temp_storage_bytes, d_input, d_output, n);

    hipMemcpy(output.data(), d_output, n * sizeof(int), hipMemcpyDeviceToHost);

    hipFree(d_input);
    hipFree(d_output);
    hipFree(d_temp_storage);
}

// Thrust implementation of exclusive scan
void exclusive_scan_thrust(const std::vector<int>& input, std::vector<int>& output) {
    thrust::host_vector<int> h_input(input.begin(), input.end());
    thrust::device_vector<int> d_input = h_input;
    thrust::device_vector<int> d_output(input.size());

    thrust::exclusive_scan(d_input.begin(), d_input.end(), d_output.begin());

    thrust::copy(d_output.begin(), d_output.end(), output.begin());
}

void calc_mean_stddev(const std::vector<double>& times, double& mean, double& stddev) {
    mean = std::accumulate(times.begin(), times.end(), 0.0) / times.size();
    double sq_sum = std::inner_product(times.begin(), times.end(), times.begin(), 0.0);
    stddev = std::sqrt(sq_sum / times.size() - mean * mean);
}

int main() {
    std::vector<int> exponents = {0, 1, 2, 3, 4, 5, 6, 7, 8, 9, 10, 11, 12, 13, 14, 15, 16};  // Exponents for 2^N

    // Warm-up kernel execution
    std::vector<int> warmup_input(1024, 1);
    std::vector<int> warmup_output(1024);
    exclusive_scan(warmup_input, warmup_output);

    std::cout << "N\tmedia_custom\tmedia_CUB\tmedia_thrust\n";

    for (int exp : exponents) {
        int n = 1024 * (1 << exp);  // Calculate 1024 * 2^N
        std::vector<int> input(n);
        std::vector<int> output_gpu(n);
        std::vector<int> output_cpu(n);
        std::vector<int> output_cub(n);
        std::vector<int> output_thrust(n);

        for (int i = 0; i < n; ++i) {
            input[i] = i + 1;
        }

        std::vector<double> times_custom;
        for (int i = 0; i < 10; ++i) {
            auto start = std::chrono::high_resolution_clock::now();
            exclusive_scan(input, output_gpu);
            hipDeviceSynchronize();  // Ensure kernel execution is complete
            auto end = std::chrono::high_resolution_clock::now();
            std::chrono::duration<double, std::milli> duration = end - start;
            times_custom.push_back(duration.count());
        }

        std::vector<double> times_cub;
        for (int i = 0; i < 10; ++i) {
            auto start = std::chrono::high_resolution_clock::now();
            exclusive_scan_cub(input, output_cub);
            hipDeviceSynchronize();  // Ensure kernel execution is complete
            auto end = std::chrono::high_resolution_clock::now();
            std::chrono::duration<double, std::milli> duration = end - start;
            times_cub.push_back(duration.count());
        }

        std::vector<double> times_thrust;
        for (int i = 0; i < 10; ++i) {
            auto start = std::chrono::high_resolution_clock::now();
            exclusive_scan_thrust(input, output_thrust);
            hipDeviceSynchronize();  // Ensure kernel execution is complete
            auto end = std::chrono::high_resolution_clock::now();
            std::chrono::duration<double, std::milli> duration = end - start;
            times_thrust.push_back(duration.count());
        }

        double mean_custom, stddev_custom;
        calc_mean_stddev(times_custom, mean_custom, stddev_custom);

        double mean_cub, stddev_cub;
        calc_mean_stddev(times_cub, mean_cub, stddev_cub);

        double mean_thrust, stddev_thrust;
        calc_mean_stddev(times_thrust, mean_thrust, stddev_thrust);

        std::cout << exp << "\t" << mean_custom << "\t" << mean_cub << "\t" << mean_thrust << "\n";
    }

    return 0;
}
