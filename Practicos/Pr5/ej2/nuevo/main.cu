#include "hip/hip_runtime.h"
#include "mmio.h"
#include <thrust/sort.h>
#include <thrust/device_vector.h>
#include <thrust/execution_policy.h>
#include <thrust/device_ptr.h>
#include <thrust/unique.h>
#include <thrust/iterator/counting_iterator.h>
#include <cmath>
#include <thrust/reduce.h>
#include <thrust/scan.h>
#include <thrust/for_each.h>
#include <thrust/host_vector.h>
#include <thrust/generate.h>
#include <thrust/copy.h>
#include <thrust/random.h>
#include <thrust/inner_product.h>
#include <thrust/binary_search.h>
#include <thrust/adjacent_difference.h>
#include <thrust/iterator/constant_iterator.h>
#include <thrust/iterator/counting_iterator.h>

#define WARP_PER_BLOCK 32
#define WARP_SIZE 32
#define CUDA_CHK(call) print_cuda_state(call);
#define MAX(A,B)        (((A)>(B))?(A):(B))
#define MIN(A,B)        (((A)<(B))?(A):(B))

static inline void print_cuda_state(hipError_t code){

   if (code != hipSuccess) printf("\ncuda error: %s\n", hipGetErrorString(code));
   
}
/*
int ordenar_filas(int* RowPtrL, int* ColIdxL, VALUE_TYPE *Val, int n, int* iorder) {
    // ... (previous code remains the same)

    // Crear y inicializar los vectores necesarios
    thrust::device_vector<int> d_ivects(7 * max_level, 0);
    thrust::device_vector<int> d_ivect_size(n);
    thrust::device_vector<int> d_iorder(n);

    // Get raw pointers for use in device code
    int* d_ivects_raw = thrust::raw_pointer_cast(d_ivects.data());
    unsigned int* d_niveles_raw = thrust::raw_pointer_cast(d_niveles.data());

    // Calcular los comienzos de cada par nivel-tamaño
    thrust::for_each(thrust::make_counting_iterator(0), thrust::make_counting_iterator(n), 
        [=] __device__ (int i) {
            int level = d_niveles_raw[i] - 1;
            int row_size = RowPtrL[i + 1] - RowPtrL[i] - 1;
            int size_class = (row_size == 0) ? 6 : (row_size == 1) ? 0 : (row_size <= 2) ? 1 :
                             (row_size <= 4) ? 2 : (row_size <= 8) ? 3 : (row_size <= 16) ? 4 : 5;

            atomicAdd(&d_ivects_raw[7 * level + size_class], 1);
        }
    );

    // Hacer un escaneo exclusivo para determinar los índices de inicio
    thrust::exclusive_scan(d_ivects.begin(), d_ivects.end(), d_ivects.begin());

    // Asignar filas a sus posiciones
    thrust::for_each(thrust::make_counting_iterator(0), thrust::make_counting_iterator(n), 
        [=] __device__ (int i) {
            int level = d_niveles_raw[i] - 1;
            int row_size = RowPtrL[i + 1] - RowPtrL[i] - 1;
            int size_class = (row_size == 0) ? 6 : (row_size == 1) ? 0 : (row_size <= 2) ? 1 :
                             (row_size <= 4) ? 2 : (row_size <= 8) ? 3 : (row_size <= 16) ? 4 : 5;

            int position = atomicAdd(&d_ivects_raw[7 * level + size_class], 1);
            d_iorder[position] = i;
            d_ivect_size[position] = (size_class == 6) ? 0 : pow(2, size_class);
        }
    );

    // ... (rest of the code remains the same)
}
*/
__global__ void kernel_analysis_L(const int* __restrict__ row_ptr,
	const int* __restrict__ col_idx,
	volatile int* is_solved, int n,
	unsigned int* niveles) {
	extern volatile __shared__ int s_mem[];

	if(threadIdx.x==0&&blockIdx.x==0) printf("%i\n", WARP_PER_BLOCK);
	int* s_is_solved = (int*)&s_mem[0];
	int* s_info = (int*)&s_is_solved[WARP_PER_BLOCK];

	int wrp = (threadIdx.x + blockIdx.x * blockDim.x) / WARP_SIZE;
	int local_warp_id = threadIdx.x / WARP_SIZE;

	int lne = threadIdx.x & 0x1f;

	if (wrp >= n) return;

	int row = row_ptr[wrp];
	int start_row = blockIdx.x * WARP_PER_BLOCK;
	int nxt_row = row_ptr[wrp + 1];

	int my_level = 0;
	if (lne == 0) {
		s_is_solved[local_warp_id] = 0;
		s_info[local_warp_id] = 0;
	}

	__syncthreads();

	int off = row + lne;
	int colidx = col_idx[off];
	int myvar = 0;

	while (off < nxt_row - 1)
	{
		colidx = col_idx[off];
		if (!myvar)
		{
			if (colidx > start_row) {
				myvar = s_is_solved[colidx - start_row];

				if (myvar) {
					my_level = max(my_level, s_info[colidx - start_row]);
				}
			} else
			{
				myvar = is_solved[colidx];

				if (myvar) {
					my_level = max(my_level, niveles[colidx]);
				}
			}
		}

		if (__all_sync(__activemask(), myvar)) {

			off += WARP_SIZE;
			//           colidx = col_idx[off];
			myvar = 0;
		}
	}
	__syncwarp();
	
	for (int i = 16; i >= 1; i /= 2) {
		my_level = max(my_level, __shfl_down_sync(__activemask(), my_level, i));
	}

	if (lne == 0) {

		s_info[local_warp_id] = 1 + my_level;
		s_is_solved[local_warp_id] = 1;
		niveles[wrp] = 1 + my_level;

		__threadfence();

		is_solved[wrp] = 1;
	}
}
    void CHECKcudaGetLastError(hipError_t error){
        if (error != hipSuccess) {
            fprintf(stderr, "CUDA error after Thrust operation: %s\n", hipGetErrorString(error));
        }
    }
    int* RowPtrL_d, *ColIdxL_d;
    VALUE_TYPE* Val_d;

int ordenar_filasCLAUD(int* RowPtrL, int* ColIdxL, VALUE_TYPE *Val, int n, int* iorder) {
    // Variables en el dispositivo
    thrust::device_vector<unsigned int> d_niveles(n);
    thrust::device_vector<int> d_is_solved(n);

    // Configuración de ejecución del kernel
    int num_threads = WARP_PER_BLOCK * WARP_SIZE;
    int grid = ceil((double) n * WARP_SIZE / (double) num_threads);

    thrust::fill(d_is_solved.begin(), d_is_solved.end(), 0);
    thrust::fill(d_niveles.begin(), d_niveles.end(), 0);

    // Llamada al kernel
    kernel_analysis_L<<<grid, num_threads, WARP_PER_BLOCK * (2 * sizeof(int))>>>(
        RowPtrL, ColIdxL, 
        thrust::raw_pointer_cast(d_is_solved.data()), 
        n, 
        thrust::raw_pointer_cast(d_niveles.data())
    );

    CUDA_CHK(hipDeviceSynchronize());

    // Copiar los resultados de vuelta al host
    thrust::host_vector<unsigned int> h_niveles = d_niveles;

    // Calcular el máximo nivel
    unsigned int max_level = *thrust::max_element(d_niveles.begin(), d_niveles.end());

    // Crear y inicializar los vectores necesarios
    thrust::device_vector<int> d_ivects(7 * max_level, 0);
    thrust::device_vector<int> d_ivect_size(n);
    thrust::device_vector<int> d_iorder(n);

    // Get raw pointers for use in device code
    int* d_ivects_raw = thrust::raw_pointer_cast(d_ivects.data());
    unsigned int* d_niveles_raw = thrust::raw_pointer_cast(d_niveles.data());

    // Calcular los comienzos de cada par nivel-tamaño
    thrust::for_each(thrust::make_counting_iterator(0), thrust::make_counting_iterator(n), 
        [=] __device__ (int i) {
            int level = d_niveles_raw[i] - 1;
            int row_size = RowPtrL[i + 1] - RowPtrL[i] - 1;
            int size_class = (row_size == 0) ? 6 : (row_size == 1) ? 0 : (row_size <= 2) ? 1 :
                             (row_size <= 4) ? 2 : (row_size <= 8) ? 3 : (row_size <= 16) ? 4 : 5;

            atomicAdd(&d_ivects_raw[7 * level + size_class], 1);
        }
    );

    // Hacer un escaneo exclusivo para determinar los índices de inicio
    thrust::exclusive_scan(d_ivects.begin(), d_ivects.end(), d_ivects.begin());

    // Asignar filas a sus posiciones
    thrust::for_each(thrust::make_counting_iterator(0), thrust::make_counting_iterator(n), 
        [=] __device__ (int i) {
            int level = d_niveles_raw[i] - 1;
            int row_size = RowPtrL[i + 1] - RowPtrL[i] - 1;
            int size_class = (row_size == 0) ? 6 : (row_size == 1) ? 0 : (row_size <= 2) ? 1 :
                             (row_size <= 4) ? 2 : (row_size <= 8) ? 3 : (row_size <= 16) ? 4 : 5;

            int position = atomicAdd(&d_ivects_raw[7 * level + size_class], 1);
            d_iorder[position] = i;
            d_ivect_size[position] = (size_class == 6) ? 0 : pow(2, size_class);
        }
    );

    // Copiar los resultados de vuelta al host
    thrust::copy(d_iorder.begin(), d_iorder.end(), iorder);

    // Calcular número de warps necesarios
    int n_warps = (n + WARP_SIZE - 1) / WARP_SIZE;

    return n_warps;
}
int ordenar_filas(int* RowPtrL, int* ColIdxL, VALUE_TYPE *Val, int n, int* iorder) {
    // Variables en el dispositivo
    unsigned int *d_niveles;
    int *d_is_solved;

    // Reserva de memoria en el dispositivo
    CUDA_CHK(hipMalloc((void**) &d_niveles, n * sizeof(unsigned int)));
    CUDA_CHK(hipMalloc((void**) &d_is_solved, n * sizeof(int)));

    // Configuración de ejecución del kernel
    int num_threads = WARP_PER_BLOCK * WARP_SIZE;
    int grid = ceil((double) n * WARP_SIZE / (double) num_threads);

    CUDA_CHK(hipMemset(d_is_solved, 0, n * sizeof(int)));
    CUDA_CHK(hipMemset(d_niveles, 0, n * sizeof(unsigned int)));

    // Llamada al kernel
    kernel_analysis_L<<<grid, num_threads, WARP_PER_BLOCK * (2 * sizeof(int))>>>(RowPtrL, ColIdxL, d_is_solved, n, d_niveles);
    CUDA_CHK(hipDeviceSynchronize());

    // Copiar los resultados de vuelta al host
    int *niveles = (int *) malloc(n * sizeof(int));
    CUDA_CHK(hipMemcpy(niveles, d_niveles, n * sizeof(int), hipMemcpyDeviceToHost));

    /* Paralelice a partir de aquí */
    hipError_t error = hipGetLastError();
    
    if (error != hipSuccess) {
        fprintf(stderr, "CUDA error before Thrust operation: %s\n", hipGetErrorString(error));
    }

    // Thrust operation here

    thrust::device_vector<int> d_niveles2(niveles, niveles + n);
    thrust::device_vector<int> d_ivects(7 * *thrust::max_element(d_niveles2.begin(), d_niveles2.end()));
    thrust::device_vector<int> d_ivect_size(n);
    thrust::device_vector<int> d_iorder(n);

    // Inicializar ivects a cero
    thrust::fill(d_ivects.begin(), d_ivects.end(), 0);
    error = hipGetLastError();
    if (error != hipSuccess) {
        fprintf(stderr, "CUDA error after Thrust operation: %s\n", hipGetErrorString(error));
    }
    // Calcular los comienzos de cada par nivel-tamaño
    thrust::for_each(thrust::make_counting_iterator(0), thrust::make_counting_iterator(n), [=, d_ivects_ptr = thrust::raw_pointer_cast(d_ivects.data())] __device__ (int i) {
        int level = niveles[i] - 1;
        int row_size = RowPtrL[i + 1] - RowPtrL[i] - 1;
        int size_class = (row_size == 0) ? 6 : (row_size == 1) ? 0 : (row_size <= 2) ? 1 :
                         (row_size <= 4) ? 2 : (row_size <= 8) ? 3 : (row_size <= 16) ? 4 : 5;

        atomicAdd(&d_ivects_ptr[7 * level + size_class], 1);
    });

    // Hacer un escaneo exclusivo para determinar los índices de inicio
    thrust::exclusive_scan(d_ivects.begin(), d_ivects.end(), d_ivects.begin());

    // Asignar filas a sus posiciones
    thrust::for_each(thrust::make_counting_iterator(0), thrust::make_counting_iterator(n), [=, d_ivects_ptr = thrust::raw_pointer_cast(d_ivects.data()), d_iorder_ptr = thrust::raw_pointer_cast(d_iorder.data()), d_ivect_size_ptr = thrust::raw_pointer_cast(d_ivect_size.data())] __device__ (int i) {
        int level = niveles[i] - 1;
        int row_size = RowPtrL[i + 1] - RowPtrL[i] - 1;
        int size_class = (row_size == 0) ? 6 : (row_size == 1) ? 0 : (row_size <= 2) ? 1 :
                         (row_size <= 4) ? 2 : (row_size <= 8) ? 3 : (row_size <= 16) ? 4 : 5;

        int position = atomicAdd(&d_ivects_ptr[7 * level + size_class], 1);
        d_iorder_ptr[position] = i;
        d_ivect_size_ptr[position] = (size_class == 6) ? 0 : pow(2, size_class);
    });

    // Copiar los resultados de vuelta al host para usarlos en el programa principal
    thrust::copy(d_iorder.begin(), d_iorder.end(), iorder);

    /* Termine aquí */

    // Liberación de memoria
    //d_niveles2.shrink_to_feet()
    CUDA_CHK(hipFree(d_is_solved));
    free(niveles);

    // Calcular número de warps necesarios (esto podría hacerse en el dispositivo si es necesario)
    int n_warps = 1; // Esto es solo un valor de lugar, debe calcularse correctamente

    return n_warps;
}

int main(int argc, char** argv)
{
    // report precision of floating-point
    printf("---------------------------------------------------------------------------------------------\n");
    char* precision;
    if (sizeof(VALUE_TYPE) == 4)
    {
        precision = (char*)"32-bit Single Precision";
    } else if (sizeof(VALUE_TYPE) == 8)
    {
        precision = (char*)"64-bit Double Precision";
    } else
    {
        printf("Wrong precision. Program exit!\n");
        return 0;
    }

    printf("PRECISION = %s\n", precision);


    int m, n, nnzA;
    int* csrRowPtrA;
    int* csrColIdxA;
    VALUE_TYPE* csrValA;

    int argi = 1;

    char* filename;
    if (argc > argi)
    {
        filename = argv[argi];
        argi++;
    }

    printf("-------------- %s --------------\n", filename);



    // read matrix from mtx file
    int ret_code;
    MM_typecode matcode;
    FILE* f;

    int nnzA_mtx_report;
    int isInteger = 0, isReal = 0, isPattern = 0, isSymmetric = 0;

    // load matrix
    if ((f = fopen(filename, "r")) == NULL)
        return -1;

    if (mm_read_banner(f, &matcode) != 0)
    {
        printf("Could not process Matrix Market banner.\n");
        return -2;
    }

    if (mm_is_complex(matcode))
    {
        printf("Sorry, data type 'COMPLEX' is not supported.\n");
        return -3;
    }

    char* pch, * pch1;
    pch = strtok(filename, "/");
    while (pch != NULL) {
        pch1 = pch;
        pch = strtok(NULL, "/");
    }

    pch = strtok(pch1, ".");


    if (mm_is_pattern(matcode)) { isPattern = 1; }
    if (mm_is_real(matcode)) { isReal = 1;  }
    if (mm_is_integer(matcode)) { isInteger = 1; }

    /* find out size of sparse matrix .... */
    ret_code = mm_read_mtx_crd_size(f, &m, &n, &nnzA_mtx_report);
    if (ret_code != 0)
        return -4;


    if (n != m)
    {
        printf("Matrix is not square.\n");
        return -5;
    }

    if (mm_is_symmetric(matcode) || mm_is_hermitian(matcode))
    {
        isSymmetric = 1;
        printf("input matrix is symmetric = true\n");
    } else
    {
        printf("input matrix is symmetric = false\n");
    }

    int* csrRowPtrA_counter = (int*)malloc((m + 1) * sizeof(int));
    memset(csrRowPtrA_counter, 0, (m + 1) * sizeof(int));

    int* csrRowIdxA_tmp = (int*)malloc(nnzA_mtx_report * sizeof(int));
    int* csrColIdxA_tmp = (int*)malloc(nnzA_mtx_report * sizeof(int));
    VALUE_TYPE* csrValA_tmp = (VALUE_TYPE*)malloc(nnzA_mtx_report * sizeof(VALUE_TYPE));

    /* NOTE: when reading in doubles, ANSI C requires the use of the "l"  */
    /*   specifier as in "%lg", "%lf", "%le", otherwise errors will occur */
    /*  (ANSI C X3.159-1989, Sec. 4.9.6.2, p. 136 lines 13-15)            */

    for (int i = 0; i < nnzA_mtx_report; i++)
    {
        int idxi, idxj;
        double fval;
        int ival;
        int returnvalue;

        if (isReal)
            returnvalue = fscanf(f, "%d %d %lg\n", &idxi, &idxj, &fval);
        else if (isInteger)
        {
            returnvalue = fscanf(f, "%d %d %d\n", &idxi, &idxj, &ival);
            fval = ival;
        } else if (isPattern)
        {
            returnvalue = fscanf(f, "%d %d\n", &idxi, &idxj);
            fval = 1.0;
        }

        // adjust from 1-based to 0-based
        idxi--;
        idxj--;

        csrRowPtrA_counter[idxi]++;
        csrRowIdxA_tmp[i] = idxi;
        csrColIdxA_tmp[i] = idxj;
        csrValA_tmp[i] = fval;
    }

    if (f != stdin)
        fclose(f);

    if (isSymmetric)
    {
        for (int i = 0; i < nnzA_mtx_report; i++)
        {
            if (csrRowIdxA_tmp[i] != csrColIdxA_tmp[i])
                csrRowPtrA_counter[csrColIdxA_tmp[i]]++;
        }
    }

    // exclusive scan for csrRowPtrA_counter
    int old_val, new_val;

    old_val = csrRowPtrA_counter[0];
    csrRowPtrA_counter[0] = 0;
    for (int i = 1; i <= m; i++)
    {
        new_val = csrRowPtrA_counter[i];
        csrRowPtrA_counter[i] = old_val + csrRowPtrA_counter[i - 1];
        old_val = new_val;
    }

    nnzA = csrRowPtrA_counter[m];
    csrRowPtrA = (int*)malloc((m + 1) * sizeof(int));
    memcpy(csrRowPtrA, csrRowPtrA_counter, (m + 1) * sizeof(int));
    memset(csrRowPtrA_counter, 0, (m + 1) * sizeof(int));

    csrColIdxA = (int*)malloc(nnzA * sizeof(int));
    csrValA = (VALUE_TYPE*)malloc(nnzA * sizeof(VALUE_TYPE));

    if (isSymmetric)
    {
        for (int i = 0; i < nnzA_mtx_report; i++)
        {
            if (csrRowIdxA_tmp[i] != csrColIdxA_tmp[i])
            {
                int offset = csrRowPtrA[csrRowIdxA_tmp[i]] + csrRowPtrA_counter[csrRowIdxA_tmp[i]];
                csrColIdxA[offset] = csrColIdxA_tmp[i];
                csrValA[offset] = csrValA_tmp[i];
                csrRowPtrA_counter[csrRowIdxA_tmp[i]]++;

                offset = csrRowPtrA[csrColIdxA_tmp[i]] + csrRowPtrA_counter[csrColIdxA_tmp[i]];
                csrColIdxA[offset] = csrRowIdxA_tmp[i];
                csrValA[offset] = csrValA_tmp[i];
                csrRowPtrA_counter[csrColIdxA_tmp[i]]++;
            } else
            {
                int offset = csrRowPtrA[csrRowIdxA_tmp[i]] + csrRowPtrA_counter[csrRowIdxA_tmp[i]];
                csrColIdxA[offset] = csrColIdxA_tmp[i];
                csrValA[offset] = csrValA_tmp[i];
                csrRowPtrA_counter[csrRowIdxA_tmp[i]]++;
            }
        }
    } else
    {
        for (int i = 0; i < nnzA_mtx_report; i++)
        {
            int offset = csrRowPtrA[csrRowIdxA_tmp[i]] + csrRowPtrA_counter[csrRowIdxA_tmp[i]];
            csrColIdxA[offset] = csrColIdxA_tmp[i];
            csrValA[offset] = csrValA_tmp[i];
            csrRowPtrA_counter[csrRowIdxA_tmp[i]]++;
        }
    }
 
    printf("input matrix A: ( %i, %i ) nnz = %i\n", m, n, nnzA);

    // extract L with the unit-lower triangular sparsity structure of A
    int nnzL = 0;
    int* csrRowPtrL_tmp = (int*)malloc((m + 1) * sizeof(int));
    int* csrColIdxL_tmp = (int*)malloc(nnzA * sizeof(int));
    VALUE_TYPE* csrValL_tmp = (VALUE_TYPE*)malloc(nnzA * sizeof(VALUE_TYPE));

    int nnz_pointer = 0;
    csrRowPtrL_tmp[0] = 0;
    for (int i = 0; i < m; i++)
    {
        for (int j = csrRowPtrA[i]; j < csrRowPtrA[i + 1]; j++)
        {
            if (csrColIdxA[j] < i)
            {
                csrColIdxL_tmp[nnz_pointer] = csrColIdxA[j];
                csrValL_tmp[nnz_pointer] = 1.0; //csrValA[j];
                nnz_pointer++;
            } else
            {
                break;
            }
        }

        csrColIdxL_tmp[nnz_pointer] = i;
        csrValL_tmp[nnz_pointer] = 1.0;
        nnz_pointer++;

        csrRowPtrL_tmp[i + 1] = nnz_pointer;
    }

    nnzL = csrRowPtrL_tmp[m];
    printf("A's unit-lower triangular L: ( %i, %i ) nnz = %i\n", m, n, nnzL);

    csrColIdxL_tmp = (int*)realloc(csrColIdxL_tmp, sizeof(int) * nnzL);
    csrValL_tmp = (VALUE_TYPE*)realloc(csrValL_tmp, sizeof(VALUE_TYPE) * nnzL);

    printf("---------------------------------------------------------------------------------------------\n");

    int* RowPtrL_d, *ColIdxL_d;
    VALUE_TYPE* Val_d;

    hipMalloc((void**)&RowPtrL_d, (n + 1) * sizeof(int));
    hipMalloc((void**)&ColIdxL_d, nnzL * sizeof(int));
    hipMalloc((void**)&Val_d, nnzL * sizeof(VALUE_TYPE));
  
    hipMemcpy(RowPtrL_d, csrRowPtrL_tmp, (n + 1) * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(ColIdxL_d, csrColIdxL_tmp, nnzL * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(Val_d, csrValL_tmp, nnzL * sizeof(VALUE_TYPE), hipMemcpyHostToDevice);

    int * iorder  = (int *) calloc(n,sizeof(int));

    int nwarps = ordenar_filasCLAUD(RowPtrL_d,ColIdxL_d,Val_d,n,iorder);
    //int nwarps = ordenar_filas(RowPtrL_d,ColIdxL_d,Val_d,n,iorder);

    printf("Number of warps: %i\n",nwarps);
    for(int i =0; i<n && i<20;i++)
        printf("Iorder[%i] = %i\n",i,iorder[i]);

    printf("Bye!\n");

    // done!
    free(csrColIdxA);
    free(csrValA);
    free(csrRowPtrA);

    free(csrColIdxL_tmp);
    free(csrValL_tmp);
    free(csrRowPtrL_tmp);

    return 0;
}
