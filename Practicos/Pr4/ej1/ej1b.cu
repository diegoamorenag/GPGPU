#include <iostream>
#include <hip/hip_runtime.h>

// kernel para transponer usando memoria compartida con columna dummy para evitar conflictos de bancos
__global__ void kernel(int *input, int *output, int width, int height) {
    __shared__ int tile[32][33]; // Agregar una columna dummy

    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    // Cargar datos en la memoria compartida
    if (x < width && y < height) {
        tile[threadIdx.y][threadIdx.x] = input[y * width + x];
    }
    __syncthreads();

    // Escribir datos transpuestos desde la memoria compartida a la global
    int transpos_x = blockIdx.y * blockDim.y + threadIdx.x;
    int transpos_y = blockIdx.x * blockDim.x + threadIdx.y;
    if (transpos_x < height && transpos_y < width) {
        output[transpos_y * height + transpos_x] = tile[threadIdx.x][threadIdx.y];
    }
}

void lanzador() {
    int width = 16384;
    int height = 16384;
    size_t bytes = width * height * sizeof(int);

    int *h_input, *h_output;
    int *d_input, *d_output;

    // Reserva de memoria en el host
    h_input = (int*)malloc(bytes);
    h_output = (int*)malloc(bytes);

    // Inicialización de la matriz de entrada
    for (int i = 0; i < height; i++) {
        for (int j = 0; j < width; j++) {
            h_input[i * width + j] = i * width + j;
        }
    }

    // Reserva de memoria en el device
    hipMalloc(&d_input, bytes);
    hipMalloc(&d_output, bytes);

    // Copia de datos hacia el device
    hipMemcpy(d_input, h_input, bytes, hipMemcpyHostToDevice);

    // Configuración del tamaño de bloque y de grilla
    dim3 blockSize(32, 32);
    dim3 gridSize((width + blockSize.x - 1) / blockSize.x, (height + blockSize.y - 1) / blockSize.y);

    // Eventos para medir el tiempo
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // Inicia medición
    hipEventRecord(start);

    // Lanzamiento del kernel
    kernel<<<gridSize, blockSize>>>(d_input, d_output, width, height);

    // Finaliza medición
    hipEventRecord(stop);
    hipEventSynchronize(stop);

    // Calcular el tiempo transcurrido
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    // Copia de resultados hacia el host
    hipMemcpy(h_output, d_output, bytes, hipMemcpyDeviceToHost);

    // Impresión de secciones de la matriz
    //std::cout << "Bloque de la matriz original:\n";
    //printMatrixSection(h_input, width, height, 0, 5, 0, 5);

    //std::cout << "Bloque de la matriz transpuesta:\n";
    //printMatrixSection(h_output, height, width, 0, 5, 0, 5);

    std::cout << "Tiempo de ejecucion del kernel: " << milliseconds << " ms\n";
    //std::cout << milliseconds << std::endl;

    // Liberar memoria y eventos
    hipFree(d_input);
    hipFree(d_output);
    free(h_input);
    free(h_output);
    hipEventDestroy(start);
    hipEventDestroy(stop);
}

int main() {
    for (int i = 0; i < 10; i++) {
        lanzador();
    }
    return 0;
}
