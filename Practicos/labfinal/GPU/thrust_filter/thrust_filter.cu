#include "hip/hip_runtime.h"
#include <algorithm>
#include <cmath>
#include <hip/hip_runtime.h>
#include <hip/hip_texture_types.h>
#include <>
#include <fstream>
#include <iostream>
#include <numeric>
#include <sstream>
#include <stdexcept>
#include <string>
#include <vector>
#include <thrust/sort.h>
#include <thrust/device_vector.h>
#include <thrust/execution_policy.h>

using namespace std;

struct PGMImage {
    int width;
    int height;
    int max_val;
    std::vector<unsigned char> data;
};

// Funcion para leer una imagen PGM
PGMImage readPGM(const char* filename) {
    std::ifstream file(filename, std::ios::binary);
    if (!file) {
        throw std::runtime_error("No se pudo abrir el archivo.");
    }

    PGMImage img;
    std::string line;
    std::getline(file, line);
    if (line != "P5" && line != "P2") {
        throw std::runtime_error("Formato de archivo no soportado. Solo se admite PGM binario (P5) o ASCII (P2).");
    }

    bool isBinary = (line == "P5");

    // Saltar comentarios
    while (std::getline(file, line)) {
        if (line[0] != '#') break;
    }

    std::istringstream iss(line);
    iss >> img.width >> img.height;
    file >> img.max_val;
    file.ignore(); // Saltar el carácter de nueva línea

    img.data.resize(img.width * img.height);
    if (isBinary) {
        file.read(reinterpret_cast<char*>(img.data.data()), img.data.size());
    } else {
        for (int i = 0; i < img.width * img.height; ++i) {
            int pixel;
            file >> pixel;
            img.data[i] = static_cast<unsigned char>(pixel);
        }
    }

    return img;
}

// Funcion para escribir una imagen PGM
void writePGM(const std::string& filename, const PGMImage& img) {
    std::ofstream file(filename, std::ios::binary);
    if (!file) {
        throw std::runtime_error("No se pudo crear el archivo: " + filename);
    }

    file << "P5\n" << img.width << " " << img.height << "\n" << img.max_val << "\n";
    file.write(reinterpret_cast<const char*>(img.data.data()), img.data.size());
}

template <int BLOCK_DIM_X, int BLOCK_DIM_Y, int WINDOW_SIZE>
__global__ void medianFilterSharedKernel(unsigned char* input, unsigned char* output, int width, int height) {
    __shared__ unsigned char sharedMem[(BLOCK_DIM_Y + WINDOW_SIZE - 1) * (BLOCK_DIM_X + WINDOW_SIZE - 1)];

    int tx = threadIdx.x;
    int ty = threadIdx.y;
    int bx = blockIdx.x * BLOCK_DIM_X;
    int by = blockIdx.y * BLOCK_DIM_Y;
    int x = bx + tx;
    int y = by + ty;

    int sharedWidth = BLOCK_DIM_X + WINDOW_SIZE - 1;
    int sharedHeight = BLOCK_DIM_Y + WINDOW_SIZE - 1;

    // Cargar datos en memoria compartida fila por fila
    for (int row = ty; row < sharedHeight; row += BLOCK_DIM_Y) {
        for (int col = tx; col < sharedWidth; col += BLOCK_DIM_X) {
            int globalX = bx + col - WINDOW_SIZE / 2;
            int globalY = by + row - WINDOW_SIZE / 2;

            if (globalX >= 0 && globalX < width && globalY >= 0 && globalY < height) {
                sharedMem[row * sharedWidth + col] = input[globalY * width + globalX];
            } else {
                sharedMem[row * sharedWidth + col] = 0;
            }
        }
    }
    __syncthreads();

    // Ordenar y encontrar la mediana usando thrust
    if (x < width && y < height) {
        unsigned char window[WINDOW_SIZE * WINDOW_SIZE];
        int idx = 0;
        for (int wy = 0; wy < WINDOW_SIZE; wy++) {
            for (int wx = 0; wx < WINDOW_SIZE; wx++) {
                window[idx++] = sharedMem[(ty + wy) * sharedWidth + (tx + wx)];
            }
        }

        thrust::sort(thrust::seq, window, window + WINDOW_SIZE * WINDOW_SIZE);
        output[y * width + x] = window[(WINDOW_SIZE * WINDOW_SIZE) / 2];  // Mediana
    }
}

// Funcion para aplicar el filtro de mediana en la GPU y medir el tiempo
float applyMedianFilterGPU(const PGMImage& input, PGMImage& output, int windowSize) {
    unsigned char *d_input, *d_output;
    size_t size = input.width * input.height * sizeof(unsigned char);

    hipMalloc(&d_input, size);
    hipMalloc(&d_output, size);
    hipMemcpy(d_input, input.data.data(), size, hipMemcpyHostToDevice);

    const int BLOCK_DIM_X = 32;
    const int BLOCK_DIM_Y = 32;
    dim3 blockSize(BLOCK_DIM_X, BLOCK_DIM_Y);
    dim3 gridSize((input.width + BLOCK_DIM_X - 1) / BLOCK_DIM_X, (input.height + BLOCK_DIM_Y - 1) / BLOCK_DIM_Y);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);

    // Lanzar el kernel apropiado según el tamaño de la ventana
    switch (windowSize) {
        case 3:
            medianFilterSharedKernel<BLOCK_DIM_X, BLOCK_DIM_Y, 3><<<gridSize, blockSize>>>(d_input, d_output, input.width, input.height);
            break;
        case 5:
            medianFilterSharedKernel<BLOCK_DIM_X, BLOCK_DIM_Y, 5><<<gridSize, blockSize>>>(d_input, d_output, input.width, input.height);
            break;
        case 7:
            medianFilterSharedKernel<BLOCK_DIM_X, BLOCK_DIM_Y, 7><<<gridSize, blockSize>>>(d_input, d_output, input.width, input.height);
            break;
        case 9:
            medianFilterSharedKernel<BLOCK_DIM_X, BLOCK_DIM_Y, 9><<<gridSize, blockSize>>>(d_input, d_output, input.width, input.height);
            break;
        case 11:
            medianFilterSharedKernel<BLOCK_DIM_X, BLOCK_DIM_Y, 11><<<gridSize, blockSize>>>(d_input, d_output, input.width, input.height);
            break;
        default:
            throw std::runtime_error("Tamano de ventana no soportado");
    }

    hipEventRecord(stop);
    hipEventSynchronize(stop);

    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    hipMemcpy(output.data.data(), d_output, size, hipMemcpyDeviceToHost);

    hipFree(d_input);
    hipFree(d_output);
    hipEventDestroy(start);
    hipEventDestroy(stop);

    return milliseconds;
}

int main(int argc, char* argv[]) {
    if (argc != 4) {
        std::cerr << "Uso: " << argv[0] << " <imagen_entrada.pgm> <imagen_salida.pgm> <tamaño_ventana>" << std::endl;
        return 1;
    }

    const char* inputFilename = argv[1];
    const char* outputFilename = argv[2];
    int windowSize = std::atoi(argv[3]);

    if (windowSize % 2 == 0) {
        std::cerr << "El tamano de la ventana debe ser impar." << std::endl;
        return 1;
    }

    try {
        PGMImage img = readPGM(inputFilename);
        PGMImage filtered = img; // Inicializar con la misma estructura

        const int NUM_ITERATIONS = 10;
        std::vector<float> times(NUM_ITERATIONS);

        for (int i = 0; i < NUM_ITERATIONS; ++i) {
            times[i] = applyMedianFilterGPU(img, filtered, windowSize);
        }

        // Calcular media
        float mean = std::accumulate(times.begin(), times.end(), 0.0f) / NUM_ITERATIONS;

        // Calcular desviación estándar
        float sq_sum = std::inner_product(times.begin(), times.end(), times.begin(), 0.0f);
        float stdev = std::sqrt(sq_sum / NUM_ITERATIONS - mean * mean);

        std::cout << "Tiempo promedio: " << mean << " ms" << std::endl;
        std::cout << "Desviacion estandar: " << stdev << " ms" << std::endl;

        writePGM(outputFilename, filtered);
        std::cout << "Filtro mediana aplicado exitosamente. Resultado guardado en " << outputFilename << std::endl;
    } catch (const std::exception& e) {
        std::cerr << "Error: " << e.what() << std::endl;
        return 1;
    }

    return 0;
}