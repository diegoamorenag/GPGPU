#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <>
#include <iostream>
#include <fstream>
#include <vector>
#include <algorithm>
#include <string>
#include <sstream>

#include <cmath>
#include <numeric>
#include <algorithm>

struct PGMImage {
    int width;
    int height;
    int max_val;
    std::vector<unsigned char> data;
};

// Función para leer una imagen PGM
PGMImage readPGM(const std::string& filename) {
    std::ifstream file(filename, std::ios::binary);
    if (!file) {
        throw std::runtime_error("No se pudo abrir el archivo: " + filename);
    }

    PGMImage img;
    std::string line;
    std::getline(file, line);
    if (line != "P5") {
        throw std::runtime_error("Formato de archivo no soportado. Solo se admite PGM binario (P5).");
    }

    // Saltar comentarios
    while (std::getline(file, line)) {
        if (line[0] != '#') break;
    }

    std::istringstream iss(line);
    iss >> img.width >> img.height;
    file >> img.max_val;
    file.ignore(); // Saltar el carácter de nueva línea

    img.data.resize(img.width * img.height);
    file.read(reinterpret_cast<char*>(img.data.data()), img.data.size());

    return img;
}

// Función para escribir una imagen PGM
void writePGM(const std::string& filename, const PGMImage& img) {
    std::ofstream file(filename, std::ios::binary);
    if (!file) {
        throw std::runtime_error("No se pudo crear el archivo: " + filename);
    }

    file << "P5\n" << img.width << " " << img.height << "\n" << img.max_val << "\n";
    file.write(reinterpret_cast<const char*>(img.data.data()), img.data.size());
}

// Radix sort implementation for unsigned char (8-bit integers)
__device__ void radixSort(unsigned char* arr, int n) {
    unsigned char output[256];  // Assuming window size is at most 16x16 = 256
    int count[256] = {0};

    // Count occurrences of each digit
    for (int i = 0; i < n; i++) {
        count[arr[i]]++;
    }

    // Compute cumulative count
    for (int i = 1; i < 256; i++) {
        count[i] += count[i - 1];
    }

    // Build the output array
    for (int i = n - 1; i >= 0; i--) {
        output[count[arr[i]] - 1] = arr[i];
        count[arr[i]]--;
    }

    // Copy the output array to original array
    for (int i = 0; i < n; i++) {
        arr[i] = output[i];
    }
}
__device__ unsigned int getBit(unsigned char value, int bitPosition) {
    return (value >> bitPosition) & 1;
}

__global__ void computeFlags(unsigned char* input, int* flags, int n, int bitPosition) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n) {
        flags[idx] = !getBit(input[idx], bitPosition);
    }
}

__global__ void radixSortStep(unsigned char* input, unsigned char* output, int* prefixSum, int n, int bitPosition) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n) {
        unsigned int bit = getBit(input[idx], bitPosition);
        int position;
        if (bit == 0) {
            position = prefixSum[idx];
        } else {
            position = idx - prefixSum[idx] + prefixSum[n-1];
        }
        output[position] = input[idx];
    }
}

__global__ void checkIfSorted(unsigned char* input, int* isSorted, int n) {
    __shared__ int localIsSorted;
    if (threadIdx.x == 0) {
        localIsSorted = 1;
    }
    __syncthreads();

    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n - 1) {
        if (input[idx] > input[idx + 1]) {
            atomicAnd(&localIsSorted, 0);
        }
    }
    __syncthreads();

    if (threadIdx.x == 0) {
        atomicAnd(isSorted, localIsSorted);
    }
}

void radixSort(unsigned char* d_input, unsigned char* d_output, int n) {
    int* d_flags;
    int* d_prefixSum;
    int* d_isSorted;
    hipMalloc(&d_flags, n * sizeof(int));
    hipMalloc(&d_prefixSum, n * sizeof(int));
    hipMalloc(&d_isSorted, sizeof(int));

    int blockSize = 256;
    int gridSize = (n + blockSize - 1) / blockSize;

    for (int bit = 0; bit < 8; ++bit) {
        computeFlags<<<gridSize, blockSize>>>(d_input, d_flags, n, bit);
        thrust::exclusive_scan(thrust::device, d_flags, d_flags + n, d_prefixSum);

        radixSortStep<<<gridSize, blockSize>>>(d_input, d_output, d_prefixSum, n, bit);

        // Swap input and output
        unsigned char* temp = d_input;
        d_input = d_output;
        d_output = temp;

        // Check if sorted
        int isSorted = 1;
        hipMemcpy(d_isSorted, &isSorted, sizeof(int), hipMemcpyHostToDevice);
        checkIfSorted<<<gridSize, blockSize>>>(d_input, d_isSorted, n);
        hipMemcpy(&isSorted, d_isSorted, sizeof(int), hipMemcpyDeviceToHost);
        if (isSorted) break;
    }

    hipFree(d_flags);
    hipFree(d_prefixSum);
    hipFree(d_isSorted);
}
// Kernel for applying median filter using shared memory and radix sort
template <int BLOCK_DIM_X, int BLOCK_DIM_Y, int WINDOW_SIZE>
__global__ void medianFilterRadixSortKernel(unsigned char* input, unsigned char* output, int width, int height) {
    __shared__ unsigned char sharedMem[BLOCK_DIM_Y + WINDOW_SIZE - 1][BLOCK_DIM_X + WINDOW_SIZE - 1];

    int tx = threadIdx.x;
    int ty = threadIdx.y;
    int bx = blockIdx.x * BLOCK_DIM_X;
    int by = blockIdx.y * BLOCK_DIM_Y;
    int x = bx + tx;
    int y = by + ty;

    // Cargar datos en memoria compartida (igual que antes)
    for (int dy = ty; dy < BLOCK_DIM_Y + WINDOW_SIZE - 1; dy += BLOCK_DIM_Y) {
        for (int dx = tx; dx < BLOCK_DIM_X + WINDOW_SIZE - 1; dx += BLOCK_DIM_X) {
            int globalX = bx + dx - WINDOW_SIZE / 2;
            int globalY = by + dy - WINDOW_SIZE / 2;

            if (globalX >= 0 && globalX < width && globalY >= 0 && globalY < height) {
                sharedMem[dy][dx] = input[globalY * width + globalX];
            } else {
                sharedMem[dy][dx] = 0;
            }
        }
    }

    __syncthreads();

    // Aplicar el filtro de mediana usando Radix Sort
    if (x < width && y < height) {
        unsigned char window[WINDOW_SIZE * WINDOW_SIZE];
        int idx = 0;

        for (int wy = 0; wy < WINDOW_SIZE; wy++) {
            for (int wx = 0; wx < WINDOW_SIZE; wx++) {
                window[idx++] = sharedMem[ty + wy][tx + wx];
            }
        }

        // Aplicar Radix Sort a la ventana
        unsigned char sortedWindow[WINDOW_SIZE * WINDOW_SIZE];
        radixSort(window, sortedWindow, WINDOW_SIZE * WINDOW_SIZE);

        output[y * width + x] = sortedWindow[(WINDOW_SIZE * WINDOW_SIZE) / 2];
    }
}

// Function to apply median filter on GPU and measure time
float applyMedianFilterGPU(const PGMImage& input, PGMImage& output, int windowSize) {
    unsigned char *d_input, *d_output;
    size_t size = input.width * input.height * sizeof(unsigned char);

    hipMalloc(&d_input, size);
    hipMalloc(&d_output, size);
    hipMemcpy(d_input, input.data.data(), size, hipMemcpyHostToDevice);

    const int BLOCK_DIM_X = 16;
    const int BLOCK_DIM_Y = 16;
    dim3 blockSize(BLOCK_DIM_X, BLOCK_DIM_Y);
    dim3 gridSize((input.width + BLOCK_DIM_X - 1) / BLOCK_DIM_X, (input.height + BLOCK_DIM_Y - 1) / BLOCK_DIM_Y);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);

    // Lanzar el kernel apropiado según el tamaño de la ventana
    switch (windowSize) {
        case 3:
            medianFilterRadixSortKernel<BLOCK_DIM_X, BLOCK_DIM_Y, 3><<<gridSize, blockSize>>>(d_input, d_output, input.width, input.height);
            break;
        case 5:
            medianFilterRadixSortKernel<BLOCK_DIM_X, BLOCK_DIM_Y, 5><<<gridSize, blockSize>>>(d_input, d_output, input.width, input.height);
            break;
        case 7:
            medianFilterRadixSortKernel<BLOCK_DIM_X, BLOCK_DIM_Y, 7><<<gridSize, blockSize>>>(d_input, d_output, input.width, input.height);
            break;
        case 9:
            medianFilterRadixSortKernel<BLOCK_DIM_X, BLOCK_DIM_Y, 9><<<gridSize, blockSize>>>(d_input, d_output, input.width, input.height);
            break;
        case 11:
            medianFilterRadixSortKernel<BLOCK_DIM_X, BLOCK_DIM_Y, 11><<<gridSize, blockSize>>>(d_input, d_output, input.width, input.height);
            break;
        default:
            throw std::runtime_error("Tamaño de ventana no soportado");
    }

    hipEventRecord(stop);
    hipEventSynchronize(stop);

    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    hipMemcpy(output.data.data(), d_output, size, hipMemcpyDeviceToHost);

    hipFree(d_input);
    hipFree(d_output);
    hipEventDestroy(start);
    hipEventDestroy(stop);

    return milliseconds;
}

int main(int argc, char* argv[]) {
    if (argc != 4) {
        std::cerr << "Uso: " << argv[0] << " <imagen_entrada.pgm> <imagen_salida.pgm> <tamaño_ventana>" << std::endl;
        return 1;
    }

    const char* inputFilename = argv[1];
    const char* outputFilename = argv[2];
    int windowSize = std::atoi(argv[3]);

    if (windowSize % 2 == 0) {
        std::cerr << "El tamaño de la ventana debe ser impar." << std::endl;
        return 1;
    }

    try {
        PGMImage img = readPGM(inputFilename);
        PGMImage filtered = img; // Inicializar con la misma estructura

        const int NUM_ITERATIONS = 100;
        std::vector<float> times(NUM_ITERATIONS);

        for (int i = 0; i < NUM_ITERATIONS; ++i) {
            times[i] = applyMedianFilterGPU(img, filtered, windowSize);
        }

        // Calcular media
        float mean = std::accumulate(times.begin(), times.end(), 0.0f) / NUM_ITERATIONS;

        // Calcular desviación estándar
        float sq_sum = std::inner_product(times.begin(), times.end(), times.begin(), 0.0f);
        float stdev = std::sqrt(sq_sum / NUM_ITERATIONS - mean * mean);

        std::cout << "Tiempo promedio: " << mean << " ms" << std::endl;
        std::cout << "Desviación estándar: " << stdev << " ms" << std::endl;

        writePGM(outputFilename, filtered);
        std::cout << "Filtro mediana aplicado exitosamente. Resultado guardado en " << outputFilename << std::endl;
    } catch (const std::exception& e) {
        std::cerr << "Error: " << e.what() << std::endl;
        return 1;
    }

    return 0;
}