#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <>
#include <iostream>
#include <fstream>
#include <vector>
#include <algorithm>
#include <string>
#include <sstream>

#include <cmath>
#include <numeric>
#include <algorithm>

// Declaración de la textura
texture<unsigned char, 2, hipReadModeElementType> texInput;

struct PGMImage {
    int width;
    int height;
    int max_val;
    std::vector<unsigned char> data;
};

// Función para leer una imagen PGM
PGMImage readPGM(const std::string& filename) {
    std::ifstream file(filename, std::ios::binary);
    if (!file) {
        throw std::runtime_error("No se pudo abrir el archivo: " + filename);
    }

    PGMImage img;
    std::string line;
    std::getline(file, line);
    if (line != "P5") {
        throw std::runtime_error("Formato de archivo no soportado. Solo se admite PGM binario (P5).");
    }

    // Saltar comentarios
    while (std::getline(file, line)) {
        if (line[0] != '#') break;
    }

    std::istringstream iss(line);
    iss >> img.width >> img.height;
    file >> img.max_val;
    file.ignore(); // Saltar el carácter de nueva línea

    img.data.resize(img.width * img.height);
    file.read(reinterpret_cast<char*>(img.data.data()), img.data.size());

    return img;
}

// Función para escribir una imagen PGM
void writePGM(const std::string& filename, const PGMImage& img) {
    std::ofstream file(filename, std::ios::binary);
    if (!file) {
        throw std::runtime_error("No se pudo crear el archivo: " + filename);
    }

    file << "P5\n" << img.width << " " << img.height << "\n" << img.max_val << "\n";
    file.write(reinterpret_cast<const char*>(img.data.data()), img.data.size());
}

// Radix sort implementation for unsigned char (8-bit integers)
__device__ void radixSort(unsigned char* arr, int n) {
    unsigned char output[256];  // Assuming window size is at most 16x16 = 256
    int count[256] = {0};

    // Count occurrences of each digit
    for (int i = 0; i < n; i++) {
        count[arr[i]]++;
    }

    // Compute cumulative count
    for (int i = 1; i < 256; i++) {
        count[i] += count[i - 1];
    }

    // Build the output array
    for (int i = n - 1; i >= 0; i--) {
        output[count[arr[i]] - 1] = arr[i];
        count[arr[i]]--;
    }

    // Copy the output array to original array
    for (int i = 0; i < n; i++) {
        arr[i] = output[i];
    }
}


template <int BLOCK_DIM_X, int BLOCK_DIM_Y, int WINDOW_SIZE>
__global__ void medianFilterOptimizedKernel(unsigned char* output, int width, int height) {
    int tx = threadIdx.x;
    int ty = threadIdx.y;
    int x = blockIdx.x * BLOCK_DIM_X + tx;
    int y = blockIdx.y * BLOCK_DIM_Y + ty;

    if (x < width && y < height) {
        unsigned char window[WINDOW_SIZE * WINDOW_SIZE];
        int idx = 0;

        for (int wy = -WINDOW_SIZE/2; wy <= WINDOW_SIZE/2; wy++) {
            for (int wx = -WINDOW_SIZE/2; wx <= WINDOW_SIZE/2; wx++) {
                float u = x + wx + 0.5f;
                float v = y + wy + 0.5f;
                window[idx++] = tex2D(texInput, u, v);
            }
        }

        radixSort(window, WINDOW_SIZE * WINDOW_SIZE);
        output[y * width + x] = window[(WINDOW_SIZE * WINDOW_SIZE) / 2];
    }
}

// Function to apply median filter on GPU and measure time
float applyMedianFilterGPU(const PGMImage& input, PGMImage& output, int windowSize) {
    unsigned char *d_output;
    size_t size = input.width * input.height * sizeof(unsigned char);

    // Allocate CUDA array and copy input data
    hipArray* cuArray;
    hipChannelFormatDesc channelDesc = hipCreateChannelDesc<unsigned char>();
    hipMallocArray(&cuArray, &channelDesc, input.width, input.height);
    hipMemcpyToArray(cuArray, 0, 0, input.data.data(), size, hipMemcpyHostToDevice);

    // Set texture parameters
    texInput.addressMode[0] = hipAddressModeClamp;
    texInput.addressMode[1] = hipAddressModeClamp;
    texInput.filterMode = hipFilterModePoint;
    texInput.normalized = false;

    // Bind the array to the texture
    hipBindTextureToArray(texInput, cuArray);

    hipMalloc(&d_output, size);

    const int BLOCK_DIM_X = 16;
    const int BLOCK_DIM_Y = 16;
    dim3 blockSize(BLOCK_DIM_X, BLOCK_DIM_Y);
    dim3 gridSize((input.width + BLOCK_DIM_X - 1) / BLOCK_DIM_X, (input.height + BLOCK_DIM_Y - 1) / BLOCK_DIM_Y);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);

    // Launch appropriate kernel based on window size
    switch (windowSize) {
        case 3:
            medianFilterOptimizedKernel<BLOCK_DIM_X, BLOCK_DIM_Y, 3><<<gridSize, blockSize>>>(d_output, input.width, input.height);
            break;
        case 5:
            medianFilterOptimizedKernel<BLOCK_DIM_X, BLOCK_DIM_Y, 5><<<gridSize, blockSize>>>(d_output, input.width, input.height);
            break;
        case 7:
            medianFilterOptimizedKernel<BLOCK_DIM_X, BLOCK_DIM_Y, 7><<<gridSize, blockSize>>>(d_output, input.width, input.height);
            break;
        case 9:
            medianFilterOptimizedKernel<BLOCK_DIM_X, BLOCK_DIM_Y, 9><<<gridSize, blockSize>>>(d_output, input.width, input.height);
            break;
        case 11:
            medianFilterOptimizedKernel<BLOCK_DIM_X, BLOCK_DIM_Y, 11><<<gridSize, blockSize>>>(d_output, input.width, input.height);
            break;
        default:
            throw std::runtime_error("Unsupported window size");
    }

    hipEventRecord(stop);
    hipEventSynchronize(stop);

    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    hipMemcpy(output.data.data(), d_output, size, hipMemcpyDeviceToHost);

    // Cleanup
    hipUnbindTexture(texInput);
    hipFreeArray(cuArray);
    hipFree(d_output);
    hipEventDestroy(start);
    hipEventDestroy(stop);

    // Check for CUDA errors
    hipError_t error = hipGetLastError();
    if (error != hipSuccess) {
        std::cerr << "CUDA error: " << hipGetErrorString(error) << std::endl;
        return -1.0f;  // Indicate error
    }

    return milliseconds;
}

int main(int argc, char* argv[]) {
    if (argc != 4) {
        std::cerr << "Uso: " << argv[0] << " <imagen_entrada.pgm> <imagen_salida.pgm> <tamaño_ventana>" << std::endl;
        return 1;
    }

    const char* inputFilename = argv[1];
    const char* outputFilename = argv[2];
    int windowSize = std::atoi(argv[3]);

    if (windowSize % 2 == 0) {
        std::cerr << "El tamaño de la ventana debe ser impar." << std::endl;
        return 1;
    }

    try {
        PGMImage img = readPGM(inputFilename);
        PGMImage filtered = img; // Inicializar con la misma estructura

        const int NUM_ITERATIONS = 100;
        std::vector<float> times(NUM_ITERATIONS);

        for (int i = 0; i < NUM_ITERATIONS; ++i) {
            times[i] = applyMedianFilterGPU(img, filtered, windowSize);
        }

        // Calcular media
        float mean = std::accumulate(times.begin(), times.end(), 0.0f) / NUM_ITERATIONS;

        // Calcular desviación estándar
        float sq_sum = std::inner_product(times.begin(), times.end(), times.begin(), 0.0f);
        float stdev = std::sqrt(sq_sum / NUM_ITERATIONS - mean * mean);

        std::cout << "Tiempo promedio: " << mean << " ms" << std::endl;
        std::cout << "Desviación estándar: " << stdev << " ms" << std::endl;

        writePGM(outputFilename, filtered);
        std::cout << "Filtro mediana aplicado exitosamente. Resultado guardado en " << outputFilename << std::endl;
    } catch (const std::exception& e) {
        std::cerr << "Error: " << e.what() << std::endl;
        return 1;
    }

    return 0;
}